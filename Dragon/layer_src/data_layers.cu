#include "layer_include/data_layers.hpp"
template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top){
	Batch<Dtype> *batch = full.pop("DataLayer prefectching queue is now empty");
	top[0]->reshapeLike(batch->data);
	dragon_gpu_copy(batch->data.count(),top[0]->mutable_gpu_data(), batch->data.gpu_data());

	if (has_labels){
		top[1]->reshapeLike(batch->label);
		dragon_gpu_copy(batch->label.count(), top[1]->mutable_gpu_data(), batch->label.gpu_data());
	}
	// Ensure the copy is synchronous wrt the host, so that the next batch isn't
	// copied in meanwhile.
	CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
	free.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

